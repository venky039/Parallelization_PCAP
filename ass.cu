#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define X 1
#define EMPTY 10
#define NO_WINNER 20
#define COLOR_RED "\x1b[31m"
#define COLOR_GREEN "\x1b[32m"
#define COLOR_YELLOW "\x1b[33m"
#define COLOR_BLUE "\x1b[34m"
#define COLOR_CYAN "\x1b[36m"
#define COLOR_RESET "\x1b[0m"

#define N 3
#define M N
typedef unsigned char symbol_t;

typedef struct board {
    symbol_t m[N][M];
    unsigned short n_empty;
} board_t;

typedef struct move {
    unsigned short i, j;
} move_t;

typedef struct job_struct {
    int alpha;
    symbol_t symbol;
    board_t board;
}Sjob;








board_t* create_board() {
    int i, j;
    board_t* board = (board_t*) malloc(sizeof(board_t));
    for(i = 0; i < N; i++) {
        for(j = 0; j < M; j++) {
            board->m[i][j] = EMPTY;
        }
    }
    board->n_empty = N * M;

    return board;
}

__device__ __host__ void put_symbol(board_t* board, symbol_t symbol, move_t* move) {
    board->m[move->i][move->j] = symbol;
    board->n_empty --;
}

__device__ __host__ void clear_symbol(board_t* board, move_t* move) {
    board->m[move->i][move->j] = EMPTY;
    board->n_empty ++;
}

__device__ __host__  symbol_t winner(board_t* b) {
    int i, j;
    symbol_t sym;
    int equal;

    // check on lines
    for(i = 0; i < N; i++) {
        equal = 1;
        sym = b->m[i][0];
        if(sym != EMPTY) {
            for(j = 1; j < M; j++) {
                if(b->m[i][j] != sym) {
                    equal = 0;
                    break;
                }
            }

            if(equal == 1) {
                return sym;
            }
        }
   
    }

    // check on columns
    for(i = 0; i < M; i++) {
        equal = 1;
        sym = b->m[0][i];
        if(sym != EMPTY) {
            for(j = 1; j < N; j++) {
                if(b->m[j][i] != sym) {
                    equal = 0;
                    break;
                }
            }

            if(equal == 1) {
                return sym;
            }
        }
   
    }
   
    // main diagonal   
    equal = 1;
    sym = b->m[0][0];
    if(sym != EMPTY) {
        for(i = 1; i < N; i++) {
            if(b->m[i][i] != sym) {
                equal = 0;
                break;
            }
        }

        if(equal == 1) {
            return sym;
        }
    }

    // secondary diagonal
    equal = 1;
    sym = b->m[0][M-1];
    if(sym != EMPTY) {
        for(i = 1; i < N; i++) {
            if(b->m[i][M-i-1] != sym) {
                equal = 0;
                break;
            }
        }

        if(equal == 1) {
            return sym;
        }
    }

    if(b->n_empty == 0) {
        return NO_WINNER;
    }

    return EMPTY;
}

void print_board(board_t* board) {
    int i, j;
    for(i = 0; i < N; i++) {
        printf("\t\t");
        for(j = 0; j < M; j++) {
            if(board->m[i][j] == X) {
                printf(COLOR_YELLOW"   X   ");
            } else if(board->m[i][j] == 0) {
                printf(COLOR_YELLOW"   O   ");
            } else {
                printf(COLOR_YELLOW"   -   ");
            }
            if(j<M-1)
            printf(COLOR_YELLOW"|");
        }
        printf(COLOR_YELLOW"\n\t\t  -------------------\n");
    }
}

void print_board_player(board_t* board) {
    int i, j;
    int qw=1;
    for(i = 0; i < N; i++) {
        printf("\t\t");
        for(j = 0; j < M; j++) {
            if(board->m[i][j] == X) {
                printf(COLOR_YELLOW"   X   ");
            } else if(board->m[i][j] == 0) {
                printf(COLOR_YELLOW"   Y   ");
            } else {
                printf(COLOR_YELLOW"   %d   ",qw++);
            }
            if(j<M-1)
            printf(COLOR_YELLOW"|");
        }
        printf(COLOR_YELLOW"\n\t\t  -------------------\n");
    }
}

 __device__ __host__  move_t** get_all_possible_moves(board_t* board, symbol_t symbol, int* n) {
    int i,j;

    move_t** list = (move_t**) malloc(board->n_empty * sizeof(move_t*));
    *n = 0;

    for(i = 0; i < N; i++) {
        for(j = 0; j < M; j++) {
            if(board->m[i][j] == EMPTY) {
                list[(*n)] = (move_t*) malloc(sizeof(move_t));
                list[(*n)]->i = i;
                list[(*n)]->j = j;
                (*n) ++;
            }
        }

    }
    return list;
}

__device__ __host__  symbol_t other_symbol(symbol_t symbol) {
    return 1 - symbol;
}



  

__device__ __host__ int get_score(board_t* board, int depth, symbol_t symbol) {
    symbol_t result = winner(board);
   
    if(result == symbol) {
        return N * M + 10 - depth;
    } else if(result != EMPTY && result != NO_WINNER) {
        return -(N * M) - 10 + depth;
    } else if(result == NO_WINNER) {
        return 1;
    }

    return 0;
}

__device__ __host__ int move(board_t* board, symbol_t symbol, int depth, int alpha, int beta) {
    int n, i;
    move_t* max_move;
    int score = get_score(board, depth, symbol);

    if(score != 0) {
        return score;
    }

    move_t** moves = get_all_possible_moves(board, symbol, &n);
    for(i = 0; i < n; i++) {
        put_symbol(board, symbol, moves[i]);
        score = -move(board, other_symbol(symbol), depth + 1, -beta, -alpha);
        clear_symbol(board, moves[i]);

        if(score > alpha) {
            alpha = score;
            max_move = moves[i];
        }

        if(alpha >= beta) {
            break;
        }
    }

    for(i = 0; i < n; i++) {
        free(moves[i]);
    }

    free(moves);
   
    return alpha;
}
__global__ void GetScoreKernel(Sjob *a,int* sc) {
    
    int ci = threadIdx.x;
    sc[ci] = -move(&(a[ci].board), a[ci].symbol, 0, -9999, -(a[ci].alpha));

}


        int main()
        {

              Sjob* d_jobs;
            int * d_scores;
            symbol_t result;
    symbol_t current_symbol = X;
    board_t* board = create_board();
    int score;
  //  symbol_t done_symbol = 2;
            int  n, best_score_index, best_score;
        move_t** moves;

        int current_move[100];

       

struct job_struct* job = (Sjob *)malloc(sizeof(struct job_struct));
        Sjob jobs[200];

        while(1)
        {
            best_score = -9999;
            for(int i=0;i<200;i++)
            jobs[i].alpha = best_score;

           if(current_symbol==0)
            printf(COLOR_RESET"\t\tCPU to move \n");
        else
            printf(COLOR_RESET"\t\tPlayer to move \n");

            moves = get_all_possible_moves(board, current_symbol, &n);
            

            if((int) current_symbol==0)
            {
            if(n==0){printf(COLOR_RED"\t\tDraw\n No more Moves Left\n");exit(0);}
               

            // pass one task to each available process
            for(int i = 0; i < n; i++)
            {
              //  printf("send move %i to %i\n", i, i + 1);

                put_symbol(board, current_symbol, moves[i]);

                jobs[i].board = *board;
                jobs[i].symbol = other_symbol(current_symbol);
               
                
                clear_symbol(board, moves[i]);

                current_move[i+1] = i;
            }

            // if there are more moves to make than processes
          

            hipMalloc((void **)&d_jobs,100*sizeof(Sjob));
            hipMalloc((void **)&d_scores,100*sizeof(int));
            hipMemcpy(d_jobs,jobs,n*sizeof(Sjob),hipMemcpyHostToDevice);

            GetScoreKernel<<<1,n>>>(d_jobs,d_scores);
            int scores[n];
            hipMemcpy(scores,d_scores,n*sizeof(int),hipMemcpyDeviceToHost);


            

            // wait for the rest of results
            for(int i = 0; i < n; i++) {
          
                if(scores[i] > best_score) {
                    best_score = scores[i];
                    best_score_index = i;
                }
              //  printf("received score %i from %i\n", scores[i], i);
            }

            put_symbol(board, current_symbol, moves[best_score_index]);

            print_board(board);

            for(int i = 0; i < n; i++) {
                free(moves[i]);
            }

            free(moves);

            result = winner(board);
            if(result != EMPTY) {
                break;
            }
           
            }
            else
            {
                int playMove;
                print_board_player(board);
                printf(COLOR_RESET"enter move accordingly ");
                scanf("%d",&playMove);
                    put_symbol(board, current_symbol, moves[playMove-1]);

                    print_board(board);

            }
            current_symbol = 1 - current_symbol;

        }

        

        if(result==0)
        {
            printf(COLOR_GREEN"\t\tCPU Wins\n");
            exit(0);
        }
    else
        if(result==1)
            {
                printf(COLOR_GREEN"\t\tPlayer Wins\n");
                exit(0);

            }




        }

